#include "hip/hip_runtime.h"
#include"roll.h"
#include<algorithm>
#include<math.h>
#include<>

__host__ __forceinline__ void copyData(vector<int>&v, int*& tmp, int len) {
	for (int i = 0; i < len; i++)
		v[i] = tmp[i];
}

namespace nvinfer1 {
	roll::roll(const std::vector<int>& vshift_sizes, const std::vector<int>& vdims,
		const std::vector<int>& vstrids, const std::vector<int>& vshapes) {
		N = vshift_sizes.size();
		sN = vshapes.size();
		rshift_sizes = vshift_sizes;
		rdims = vdims;
		rstrids = vstrids;
		rshapes = vshapes;
		CUDA_CHECK(hipMalloc(&shifts, vshift_sizes.size() * sizeof(int)));
		CUDA_CHECK(hipMemcpy(shifts, vshift_sizes.data(), vshift_sizes.size() * sizeof(int), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMalloc(&dims, vdims.size() * sizeof(int)));
		CUDA_CHECK(hipMemcpy(dims, vdims.data(), vdims.size() * sizeof(int), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMalloc(&strides, vstrids.size() * sizeof(int)));
		CUDA_CHECK(hipMemcpy(strides, vstrids.data(), vstrids.size() * sizeof(int), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMalloc(&shapes, vshapes.size() * sizeof(int)));
		CUDA_CHECK(hipMemcpy(shapes, vshapes.data(), vshapes.size() * sizeof(int), hipMemcpyHostToDevice));
	}
	roll::~roll() {
		CUDA_CHECK(hipFree(shifts));
		CUDA_CHECK(hipFree(dims));
		CUDA_CHECK(hipFree(strides));
		CUDA_CHECK(hipFree(shapes));
	}
	// �����л�
	roll::roll(const void* data, size_t length) {
		const char* d = reinterpret_cast<const char*>(data), *a = d;
		Tn::read(d, mInputSize);
		Tn::read(d, N);
		Tn::read(d, sN);
		int size = (int)N * sizeof(int);
		rshift_sizes.resize(N);
		memcpy(rshift_sizes.data(), d, size);

		d += size;
		rdims.resize(N);
		memcpy(rdims.data(), d, size);

		std::cout << std::endl;
		d += size;

		size = (int)sN * sizeof(int);
		rstrids.resize(sN);
		memcpy(rstrids.data(), d, size);
		d += size;
		rshapes.resize(sN);
		memcpy(rshapes.data(), d, size);
		d += size;

		CUDA_CHECK(hipMalloc(&shifts, rshift_sizes.size() * sizeof(int)));
		CUDA_CHECK(hipMemcpy(shifts, rshift_sizes.data(), rshift_sizes.size() * sizeof(int), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMalloc(&dims, rdims.size() * sizeof(int)));
		CUDA_CHECK(hipMemcpy(dims, rdims.data(), rdims.size() * sizeof(int), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMalloc(&strides, rstrids.size() * sizeof(int)));
		CUDA_CHECK(hipMemcpy(strides, rstrids.data(), rstrids.size() * sizeof(int), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMalloc(&shapes, rshapes.size() * sizeof(int)));
		CUDA_CHECK(hipMemcpy(shapes, rshapes.data(), rshapes.size() * sizeof(int), hipMemcpyHostToDevice));

		assert(d == a + length);
	}
	// ���л�
	void roll::serialize(void* buffer) const {
		char* d = static_cast<char*>(buffer), *a = d;
		Tn::write(d, mInputSize);
		Tn::write(d, N);
		Tn::write(d, sN);
		int size = rshift_sizes.size() * sizeof(int);
		memcpy(d, rshift_sizes.data(), size);
		d += size;
		size = rdims.size() * sizeof(int);
		memcpy(d, rdims.data(), size);
		d += size;
		size = rstrids.size() * sizeof(int);
		memcpy(d, rstrids.data(), size);
		d += size;
		size = rshapes.size() * sizeof(int);
		memcpy(d, rshapes.data(), size);
		d += size;
		assert(d == a + getSerializationSize());
	}
	size_t roll::getSerializationSize() const {
		return sizeof(mInputSize) + sizeof(N) + sizeof(sN) + rshift_sizes.size() * sizeof(int) +
			rdims.size() * sizeof(int)+ rstrids.size() * sizeof(int)+
			rshapes.size() * sizeof(int);
	}

	int roll::initialize() {
		return 0;
	}

	Dims roll::getOutputDimensions(int index, const Dims* inputs, int nbInputDims) {
		assert(nbInputDims == 1);
		Dims outputDims;
		outputDims.nbDims = inputs[0].nbDims;
		for (int i = 0; i < inputs[0].nbDims; i++)
		{
			outputDims.d[i] = inputs[0].d[i];
		}
		return outputDims;
	}

	// Set plugin namespace
	void roll::setPluginNamespace(const char* pluginNamespace)
	{
		mPluginNamespace = pluginNamespace;
	}

	const char* roll::getPluginNamespace() const
	{
		return mPluginNamespace;
	}

	// Return the DataType of the plugin output at the requested index
	DataType roll::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
	{
		return DataType::kFLOAT;
	}

	// Return true if output tensor is broadcast across a batch.
	bool roll::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const
	{
		return false;
	}

	// Return true if plugin can use input that is broadcast across batch without replication.
	bool roll::canBroadcastInputAcrossBatch(int inputIndex) const
	{
		return false;
	}

	void roll::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput)
	{

		mInputSize = 1;
		for (int i = 0; i < in[0].dims.nbDims; i++) {
			mInputSize *= in[0].dims.d[i];
		}
	}

	// Attach the plugin object to an execution context and grant the plugin the access to some context resource.
	void roll::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator)
	{
	}

	// Detach the plugin object from its execution context.
	void roll::detachFromContext() {}

	const char* roll::getPluginType() const
	{
		return "rollLayer_TRT";
	}

	const char* roll::getPluginVersion() const
	{
		return "1";
	}

	void roll::destroy()
	{
		delete this;
	}

	// Clone the plugin
	IPluginV2IOExt* roll::clone() const
	{
		roll *p = new roll(rshift_sizes, rdims, rstrids, rshapes);
		p->setPluginNamespace(mPluginNamespace);
		p->setInputSize(mInputSize);
		return p;
	}

	__global__ void rollKernel(const float *in, float *out, int size,int Ndims,const int* rshift,
		const int* rdims,const int* rstrids,const int* rshapes) {
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx >= size) return;
		int new_dim = 0;
		int new_idx = idx;;
		#pragma unroll
		for (size_t i = 0; i < Ndims; i++)
		{
			int ind = rdims[i];
			new_dim = (idx / rstrids[ind])%rshapes[ind]+rshift[i];
			//��Ҫ���� Խ��ѭ��
			if (new_dim>=rshapes[ind]) 
				new_idx += (rshift[i] - rshapes[ind])*rstrids[ind];
			else
				new_idx += rshift[i]*rstrids[ind];
		}
		out[new_idx] = in[idx];
	}
	void roll::forwardGpu(const float *const * inputs, float * output, hipStream_t stream, int batchSize) {
		int numElem = batchSize * mInputSize;
		rollKernel << <(numElem + mThreadCount - 1) / mThreadCount, mThreadCount >> > 
			(inputs[0], output, numElem, N, (const int*)shifts, (const int*)dims, (const int*)strides, (const int*)shapes);
	}

	int roll::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
	{
		forwardGpu((const float *const *)inputs, (float*)outputs[0], stream, batchSize);
		return 0;
	}

	PluginFieldCollection rollCreator::mFC{};
	std::vector<PluginField> rollCreator::mPluginAttributes;

	rollCreator::rollCreator()
	{
		mPluginAttributes.clear();
		mFC.nbFields = mPluginAttributes.size();
		mFC.fields = mPluginAttributes.data();
	}

	const char* rollCreator::getPluginName() const
	{
		return "rollLayer_TRT";
	}

	const char* rollCreator::getPluginVersion() const
	{
		return "1";
	}

	const PluginFieldCollection* rollCreator::getFieldNames()
	{
		return &mFC;
	}


	IPluginV2IOExt* rollCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
	{
		const PluginField* fields = fc->fields;
		std::vector<int> vshift_sizes, vdims,
			vstrids, vshapes;

		for (int i = 0; i < fc->nbFields; i++)
		{
			int* tmp = (int*)(fields[i].data);
			if (strcmp(fields[i].name, "shift_sizes") == 0) {
				for (int j = 0; j < fields[i].length; j++)
					vshift_sizes.push_back(tmp[j]);
			}
			else if (strcmp(fields[i].name, "dims") == 0) {
				for (int j = 0; j < fields[i].length; j++) {
					vdims.push_back(tmp[j]);
				}
			}
			else if (strcmp(fields[i].name, "strids") == 0) {
				for (int j = 0; j < fields[i].length; j++)
					vstrids.push_back(tmp[j]);
			}
			else {
				for (int j = 0; j < fields[i].length; j++)
					vshapes.push_back(tmp[j]);
			}
		}
		
		assert(vshift_sizes.size() > 0);
		assert(vshift_sizes.size() == vdims.size());
		roll* obj = new roll(vshift_sizes, vdims, vstrids, vshapes);
		obj->setPluginNamespace(mNamespace.c_str());
		return obj;
	}

	IPluginV2IOExt* rollCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
	{
		// This object will be deleted when the network is destroyed, which will
		roll* obj = new roll(serialData, serialLength);
		obj->setPluginNamespace(mNamespace.c_str());
		return obj;
	}
};